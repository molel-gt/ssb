#include "hip/hip_runtime.h"
#include <iostream>
#include <set>

#include <gmsh.h>
#include <algorithm>

using namespace std;
using namespace gmsh;

// default grid size
const int Nx = 101;
const int Ny = 201;
const int Nz = 101;
const int NUM_GRID = Nx * Ny * Nz;

struct Rectangle {
    int x0 = 0;
    int y0 = 0;
    int z0 = 0;
    int dx = 0;
    int dy = 0;
    int dz = 0;
};

__global__ void makeRectangles(int *voxelData, struct Rectangle *rectangles, int NX)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < NX){
        for (int i_y = 0; i_y < Ny - 2; i_y++){
            int squares[Nz];
            for (int i_z = 0; i_z < Nz; i_z++){
                int idx_p0 = idx + i_y * Ny + i_z * Ny * Nz;
                int idx_p1 = idx + 1 + i_y * Ny + i_z * Ny * Nz;
                int idx_p2 = idx + (i_y + 1) * Ny + i_z * Ny * Nz;
                int idx_p3 = idx + 1 + (i_y + 1) * Ny + i_z * Ny * Nz;
                if (voxelData[idx_p0] && voxelData[idx_p1] && voxelData[idx_p2] && voxelData[idx_p3]){
                    squares[i_z] = 1;
                }
                else {
                    squares[i_z] = 0;
                }
            }

            int partLength = 0;
            int startPos = 0;
            int rect_counter = 0;

            for (int i_z = 0; i_z < Nz - 1; i_z++){
                if (squares[i_z] && squares[i_z + 1]){
                    partLength++;
                }
                else if (squares[i_z] && !squares[i_z + 1]) {
                    int rect_idx = idx + i_y * Ny + i_z * Ny * Nz;
                    rectangles[rect_idx].x0 = idx;
                    rectangles[rect_idx].y0 = i_y;
                    rectangles[rect_idx].z0 = startPos;
                    rectangles[rect_idx].dx = 1;
                    rectangles[rect_idx].dy = 1;
                    rectangles[rect_idx].dz = partLength;
                    rect_counter ++;
                    startPos = i_z + 1;
                    partLength = 0;
                }
                else {
                    partLength = 0;
                    startPos = i_z + 1;
                    
                }
                if (i_z == Nz - 2 && partLength){
                    int rect_idx = idx + i_y * Ny + i_z * Ny * Nz;
                    rectangles[rect_idx].x0 = idx;
                    rectangles[rect_idx].y0 = i_y;
                    rectangles[rect_idx].z0 = startPos;
                    rectangles[rect_idx].dx = 1;
                    rectangles[rect_idx].dy = 1;
                    rectangles[rect_idx].dz = partLength;
                    rect_counter ++;
                    startPos = i_z + 1;
                }
            }

        } 
    }
}

void generateMesh(struct Rectangle *rectangles){
    gmsh::initialize();
    gmsh::logger::start();
    
    gmsh::model::add("porous");
    std::vector<std::pair<int, int> > solids;
    std::vector<std::pair<int, int> > ov;
    std::vector<std::vector<std::pair<int, int> > > ovv;
    int tag = 0;
    try
    {
        tag++;
        gmsh::model::occ::addBox(0, 0, 0, Nx - 1, Ny - 1, Nz - 1, tag);
    }
    catch (...){
        gmsh::logger::write("Could not create OpenCASCADE shapes!");
        return;
    }

    for (int idx = 0; idx < Nx - 1; idx++){
        for (int idy = 0; idy < Ny - 1; idy++){
            for (int idz = 0; idz < Nz - 1; idz++){
                int rect_index = idx + idy * Ny + idz * Ny * Nz;
                if (3 > rectangles[rect_index].dx > 0 && 3 > rectangles[rect_index].dy > 0 && 3 > rectangles[rect_index].dz > 0){
                    tag++;
                    printf("%d,%d,%d\n", rectangles[rect_index].dx, rectangles[rect_index].dy, rectangles[rect_index].dz);
                    gmsh::model::occ::addBox(rectangles[rect_index].x0, rectangles[rect_index].y0, rectangles[rect_index].z0,
                                            rectangles[rect_index].dx, rectangles[rect_index].dy, rectangles[rect_index].dz,
                                            tag
                                            );
                    solids.push_back({3, tag});
                }
            }
        }
    }

    gmsh::model::occ::cut({{3, 1}}, solids, ov, ovv, tag + 1);
    gmsh::model::occ::synchronize();

    gmsh::model::addPhysicalGroup(3, {tag + 1}, 1);

    double lcar1 = .1;
    gmsh::model::getEntities(ov, 0);
    gmsh::model::mesh::setSize(ov, lcar1);

    gmsh::model::mesh::generate(3);
    gmsh::logger::write("Writing mesh..");
    gmsh::write("../porous.msh");
    
    gmsh::finalize();
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
int main(int argc, char **argv)
{
    int *voxels = (int *)malloc(sizeof(int) * NUM_GRID);
    struct Rectangle *rectangles = (struct Rectangle *)malloc(sizeof(struct Rectangle) * NUM_GRID);
    int *d_voxels;
    struct Rectangle *d_rectangles;

    hipMalloc((void **)&d_voxels, sizeof(int) * NUM_GRID);
    hipMalloc((void **)&d_rectangles, sizeof(struct Rectangle) * NUM_GRID);

    voxels = (int *)malloc(sizeof(int) * NUM_GRID);
    rectangles = (struct Rectangle *)malloc(sizeof(struct Rectangle) * NUM_GRID);
    // build voxels
     for (int i = 0; i < 30; i++){
        voxels[i] = 1;
    }

    // copy inputs to device
    hipMemcpy(d_voxels, voxels, sizeof(int) * NUM_GRID, hipMemcpyHostToDevice);
    hipMemcpy(d_rectangles, rectangles, sizeof(struct Rectangle) * NUM_GRID, hipMemcpyHostToDevice);
    makeRectangles<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_voxels, d_rectangles, Nx);
    hipMemcpy(rectangles, d_rectangles, sizeof(struct Rectangle) * NUM_GRID, hipMemcpyDeviceToHost);

    // free memory on device
    hipFree(d_voxels); hipFree(d_rectangles);
    // build geometry
    // generateMesh(rectangles);
    printf("%d\n", rectangles[0].dz);
    return 0;
}